#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>

#ifndef MAX_THREADS
#define MAX_THREADS 1024
#endif
const int SQRT_MAX_THREADS = static_cast<int>(sqrt(MAX_THREADS));
const dim3 THREADS_PER_BLOCK(SQRT_MAX_THREADS, SQRT_MAX_THREADS);

__global__ void CUDAAddArrays(const int* dev_a, const int* dev_b, int* dev_c, const int arrayLength)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < arrayLength) {
        dev_c[i] = dev_a[i] + dev_b[i];
    }
}

__global__ void CUDASubtractArrays(const int* dev_a, const int* dev_b, int* dev_c, const int arrayLength)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < arrayLength) {
        dev_c[i] = dev_a[i] - dev_b[i];
    }
}

__global__ void CUDAMultiplyArrays(const int* dev_a, const int* dev_b, int* dev_c, 
    const int a_rows, const int a_columns, const int b_columns) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int column = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (row < a_rows && column < b_columns) {
        // temporary variable used to compute the result
        int h = 0;
        for (int i = 0; i < a_columns; ++i) {
            h += dev_a[row * a_columns + i] * dev_b[i * b_columns + column];
        }

        dev_c[row * b_columns + column] = h;
    }   
}

__host__ void AddArrays(const int* dev_a, const int* dev_b, int* dev_c, const int arrayLength) {
    dim3 numBlocks((arrayLength / MAX_THREADS) + (arrayLength % MAX_THREADS == 0 ? 0 : 1));
    CUDAAddArrays << < numBlocks, MAX_THREADS >> > (dev_a, dev_b, dev_c, arrayLength);
}

__host__ void SubtractArrays(const int* dev_a, const int* dev_b, int* dev_c, const int arrayLength) {
    dim3 numBlocks((arrayLength / MAX_THREADS) + (arrayLength % MAX_THREADS == 0 ? 0 : 1));
    CUDASubtractArrays << < numBlocks, MAX_THREADS >> > (dev_a, dev_b, dev_c, arrayLength);
}

__host__ void Multiply2d(const int* dev_a, const int* dev_b, int* dev_c, 
    const int a_rows, const int a_columns, const int b_columns) {
    dim3 numBlocks((b_columns / MAX_THREADS) + (b_columns % MAX_THREADS == 0 ? 0 : 1),
        (a_rows / MAX_THREADS) + (a_rows % MAX_THREADS == 0 ? 0 : 1));
    CUDAMultiplyArrays << <  numBlocks, THREADS_PER_BLOCK >> > (dev_a, dev_b, dev_c, a_rows, a_columns, b_columns);
}

__host__ void* Create(size_t sizeInBytes) {
    void* devPtr;
    hipMalloc(&devPtr, sizeInBytes);
    return devPtr;
}

__host__ void CopyDeviceToHost(void* dst, const void* src, size_t sizeInBytes) {
    hipMemcpy(dst, src, sizeInBytes, hipMemcpyDeviceToHost);
}

__host__ void CopyHostToDevice(void* dst, const void* src, size_t sizeInBytes) {
    hipMemcpy(dst, src, sizeInBytes, hipMemcpyHostToDevice);
}

__host__ void CopyDeviceToDevice(void* dst, const void* src, size_t sizeInBytes) {
    hipMemcpy(dst, src, sizeInBytes, hipMemcpyDeviceToDevice);
}

__host__ void CopyHostToHost(void* dst, const void* src, size_t sizeInBytes) {
    hipMemcpy(dst, src, sizeInBytes, hipMemcpyHostToHost);
}

// Waits till all the threads in the gpu finish doing their work
__host__ void Wait() {
    hipDeviceSynchronize();
}

__host__ void Free(void* devPtr) {
    hipError_t a = hipFree(devPtr);
}