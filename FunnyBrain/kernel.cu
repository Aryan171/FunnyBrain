﻿#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void AddArrays(const int* a, const int* b, int* c)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__host__ void add(const int* a, const int* b, int* c, const int size) {
    int* dev_a, 
        *dev_b,
        *dev_c;

    hipMalloc(&dev_a, size * sizeof(int));
    hipMalloc(&dev_b, size * sizeof(int));
    hipMalloc(&dev_c, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    
    AddArrays <<<1, size >>> (dev_a, dev_b, dev_c);

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}